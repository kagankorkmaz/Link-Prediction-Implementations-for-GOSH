#include "hip/hip_runtime.h"
#include <stdio.h>
#include "sigmoid.h"
#include "align.h"
using namespace std;
void init_sig_table(float*& sigmoid_lookup_table){
#ifdef ALIGNMENT
  sigmoid_lookup_table = new float[SIGMOID_TABLE_SIZE];
#else
  sigmoid_lookup_table =  (float*)aligned_malloc((size_t)(SIGMOID_TABLE_SIZE*sizeof(float)),(size_t)ALIGNMENT);
#endif
  for (int i =0; i<SIGMOID_TABLE_SIZE;i++){
    float x = (-SIGMOID_BOUND+float(i*SIGMOID_RESOLUTION));
    sigmoid_lookup_table[i]=1/(1+exp(-(x)));
  }
}

__device__ float fastSig(double num, float* table){
  if (num>=SIGMOID_BOUND)
    return 1;
  else if(num<-SIGMOID_BOUND)
    return 0;
  int index_in_table = (num+SIGMOID_BOUND)/SIGMOID_RESOLUTION;
  if (index_in_table > 1023 || index_in_table<0){
    printf("num %f sigmoid_bound %f sigmoid_res %f\n", num, SIGMOID_BOUND, SIGMOID_RESOLUTION);
  }
  return table[index_in_table];
}
